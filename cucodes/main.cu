
#include <hip/hip_runtime.h>
#include <iostream>

#define N_THREADS 3
#define N_BLOCKS 2147483647 

__global__ 
void hi_from_gpu(){
    while (true)
    {
        /* code */
    }
    
    //printf("Hi from GPU, from thread id %d and block id %d \n", threadIdx.x, blockIdx.x);
}

int main(){
    dim3 k;
    hi_from_gpu<<<N_BLOCKS, N_THREADS>>>();
    hipDeviceSynchronize();
    return 0;
}
